#include "hip/hip_runtime.h"
﻿#include "wb.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

// Ядро, выполняется на большом числе нитей 
__global__ void vecAdd(float* in1, float* in2, float* out, int len) {
    // Глобальный индекс нити 
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Выполнение обработки соответствующих данной нити данных
    out[idx] = in1[idx] + in2[idx];
}

int main(int argc, char** argv) {
    wbArg_t args;
    int inputLength;
    float* hostInput1;
    float* hostInput2;
    float* hostOutput;
    float* deviceInput1;
    float* deviceInput2;
    float* deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 =
        (float*)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 =
        (float*)wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float*)malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Выделите память GPU
    hipMalloc((void**)&deviceInput1, inputLength * sizeof(float));
    hipMalloc((void**)&deviceInput2, inputLength * sizeof(float));
    hipMalloc((void**)&deviceOutput, inputLength * sizeof(float));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Скопируйте память на GPU
    hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float),
        hipMemcpyHostToDevice);
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Инициализируйте размерности сетки и блоков
    dim3 blockDim = dim3(512, 1);
    dim3 gridDim(ceil(((float)inputLength) / ((float)blockDim.x)));

    wbTime_start(Compute, "Performing CUDA computation");
   
    vecAdd << <blockDim, gridDim >> > (deviceInput1, deviceInput2, deviceOutput,
        inputLength);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Скопируйте память GPU обратно на хост
    hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float),
        hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Освободите память GPU
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}